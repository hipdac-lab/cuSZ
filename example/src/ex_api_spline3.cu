#include "hip/hip_runtime.h"
/**
 * @file ex_api_spline3.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.2
 * @date 2021-06-06
 * (create) 2021-06-06 (rev) 2022-01-08
 *
 * (C) 2021 by Washington State University, Argonne National Laboratory
 *
 */

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include "ex_common.cuh"
#include "ex_common2.cuh"

#include "sp_path.cuh"
#include "wrapper/csr11.cuh"
#include "wrapper/interp_spline3.cuh"

#include <pwd.h>
#include <sys/types.h>
#include <unistd.h>

using std::cout;

using T = float;
using E = float;

// double const  eb          = 3e-3;
constexpr int FAKE_RADIUS = 0;
constexpr int radius      = FAKE_RADIUS;
// constexpr auto DEVICE      = cusz::LOC::DEVICE;

constexpr unsigned int dimx = 235, dimy = 449, dimz = 449;
constexpr dim3         xyz = dim3(dimx, dimy, dimz);
constexpr unsigned int len = dimx * dimy * dimz;

using Compressor = SparsityAwarePath::DefaultCompressor;
using Predictor  = cusz::Spline3<T, E, float>;
using SpReducer  = cusz::CSR11<T>;

bool        gpu_verify = true;
std::string fname("");

void predictor_detail(T* data, T* cmp, dim3 xyz, double eb, bool use_sp, hipStream_t stream = nullptr)
{
    auto BARRIER = [&]() {
        if (not stream) {
            CHECK_CUDA(hipDeviceSynchronize());
            printf("device sync'ed\n");
        }
        else {
            CHECK_CUDA(hipStreamSynchronize(stream));
            printf("stream sync'ed\n");
        }
    };

    Predictor predictor(xyz, true);
    SpReducer spreducer;

    T* xdata = data;

    T* anchor{nullptr};
    E* errctrl{nullptr};

    auto dbg_echo_nnz = [&]() {
        int __nnz = thrust::count_if(
            thrust::device, errctrl, errctrl + predictor.get_quant_footprint(),
            [] __device__(const T& x) { return x != 0; });
        cout << "__nnz: " << __nnz << '\n';
    };

    BYTE*  csr;
    size_t csr_nbyte;

    auto _1_allocate_workspace = [&]() {  //
        printf("_1_allocate_workspace\n");
        predictor.allocate_workspace();
    };

    auto _1_compress_time = [&]() {
        printf("_1_compress_time\n");
        predictor.construct(data, eb, radius, anchor, errctrl, stream);
        BARRIER();

        dbg_echo_nnz();
    };

    auto _1_decompress_time = [&]() {  //
        printf("_1_decompress_time\n");
        predictor.reconstruct(anchor, errctrl, eb, radius, xdata, stream);
        BARRIER();
    };

    auto _2_allocate_workspace = [&]() {
        printf("_2_allocate_workspace\n");
        predictor.allocate_workspace();
        auto spreducer_in_len = predictor.get_quant_footprint();
        spreducer.allocate_workspace(spreducer_in_len, true);
    };

    auto _2_compress_time = [&]() {
        printf("_2_compress_time\n");
        predictor.construct(data, eb, radius, anchor, errctrl, stream);
        BARRIER();

        dbg_echo_nnz();

        spreducer.gather(errctrl, predictor.get_quant_footprint(), csr, csr_nbyte, stream);
        BARRIER();
    };

    auto _2_decompress_time = [&]() {  //
        printf("_2_decompress_time\n");
        spreducer.scatter(csr, errctrl, stream);
        BARRIER();
        predictor.reconstruct(anchor, errctrl, eb, radius, xdata, stream);
        BARRIER();
    };

    // -----------------------------------------------------------------------------

    if (not use_sp) {
        _1_allocate_workspace();
        _1_compress_time();
        _1_decompress_time();
    }
    else {
        _2_allocate_workspace();
        _2_compress_time();
        _2_decompress_time();
    }

    if (gpu_verify)
        echo_metric_gpu(xdata, cmp, len);
    else
        echo_metric_cpu(xdata, cmp, len, true);
}

void compressor_detail(T* data, T* cmp, dim3 xyz, double eb, bool use_sp, hipStream_t stream = nullptr)
{
    Compressor compressor;
    BYTE*      compressed;
    size_t     compressed_len;

    auto xdata = data;

    // one-time ALLOCATION given the input size
    compressor.allocate_workspace(xyz);

    // COMPRESSION
    compressor.compress(data, eb, radius, compressed, compressed_len, stream);

    // prepare a space that hold the compressed file
    Capsule<BYTE> file(compressed_len);
    file.template alloc<cusz::LOC::HOST_DEVICE>();
    hipMemcpy(file.dptr, compressed, compressed_len, hipMemcpyDeviceToDevice);
    // clear & reuse for testing
    hipMemset(compressed, 0x0, compressed_len);

    // clear buffers of the components
    compressor.clear_buffer();

    // load the compressed "file" before decompression
    hipMemcpy(compressed, file.dptr, compressed_len, hipMemcpyDeviceToDevice);

    // DECOMPRESSION
    compressor.decompress(compressed, eb, radius, xdata, stream);

    if (gpu_verify)
        echo_metric_gpu(xdata, cmp, len);
    else
        echo_metric_cpu(xdata, cmp, len);
}

void predictor_demo(bool use_sp, double eb = 1e-2, bool use_compressor = false, bool use_r2r = false)
{
    Capsule<T> exp(len, "exp data");
    Capsule<T> bak(len, "bak data");

    cout << "using eb = " << eb << '\n';
    cout << fname << '\n';

    exp.template alloc<cusz::LOC::HOST_DEVICE>().template from_file<cusz::LOC::HOST>(fname).host2device();
    bak.template alloc<cusz::LOC::HOST_DEVICE>();
    hipMemcpy(bak.hptr, exp.hptr, len * sizeof(T), hipMemcpyHostToHost);
    bak.host2device();

    double adjusted_eb;
    figure_out_eb(exp, eb, adjusted_eb, use_r2r);

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    if (not use_compressor) {  //
        predictor_detail(exp.dptr, bak.dptr, xyz, adjusted_eb, use_sp, stream);
    }
    else {
        compressor_detail(exp.dptr, bak.dptr, xyz, adjusted_eb, use_sp, stream);
    }
    if (stream) CHECK_CUDA(hipStreamDestroy(stream));

    exp.template free<cusz::LOC::HOST_DEVICE>();
    bak.template free<cusz::LOC::HOST_DEVICE>();
}

int main(int argc, char** argv)
{
    auto help = []() {
        printf("./prog <1:select> <2:fname> [3:eb = 1e-2] [4:mode = abs] [5:verify = gpu]\n");
        printf("<..> necessary, [..] optional\n");
        printf(
            "argv[1]: "
            "(1) predictor demo, "
            "(2) predictor-spreducer demo, "
            "(3) compressor demo\n"
            "argv[2]: filename\n"
            "argv[3]: error bound (default to \"1e-2\")\n"
            "argv[4]: mode, abs or r2r (default to \"abs\")\n"
            "argv[5]: if using GPU to verify (default to \"gpu\")\n");
    };

    auto eb      = 1e-2;
    auto mode    = std::string("abs");
    auto use_r2r = false;

    if (argc < 3) {  //
        help();
    }
    else if (argc >= 3) {
        auto demo = atoi(argv[1]);
        fname     = std::string(argv[2]);
        if (argc >= 4) eb = atof(argv[3]);
        if (argc >= 5) mode = std::string(argv[4]);
        if (argc == 6) gpu_verify = std::string(argv[5]) == "gpu";
        use_r2r = mode == "r2r";

        if (demo == 1)
            predictor_demo(false, eb, false, use_r2r);
        else if (demo == 2)
            predictor_demo(true, eb, false, use_r2r);
        else if (demo == 3)
            predictor_demo(true, eb, true, use_r2r);
        else
            help();
    }

    return 0;
}
